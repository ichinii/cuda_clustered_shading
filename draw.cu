#include "hip/hip_runtime.h"
#pragma once

#include "common.cu"

#define max_dist 200.0f
#define min_dist 0.01f
#define max_it 1000
#define planes_size (grid_size + 1u)
#define planes_count (planes_size + planes_size + planes_size)

__device__ struct Scene {
    Span *spans;
    int *indices;
    KeyValue *mortons;
    Light *l;
    Plane *f;
    int lights_count;
    unsigned int tile_index;
    View v;
    Camera cam;
};

__device__ float sdmin(float a, float b) {
    return abs(a) < abs(b) ? a : b;
}

__device__ float sdf_plane(vec3 p, vec3 n) {
    return dot(p, n);
}

__device__ float sdf_sphere(vec3 p) {
    return length(p);
}

// thanks Inigo Quilez (https://iquilezles.org/articles/distfunctions/)
__device__ float sdf_capsule(vec3 p, vec3 a, vec3 b, float r) {
    vec3 pa = p - a;
    vec3 ba = b - a;
    float h = clamp(dot(pa, ba) / dot(ba, ba), 0.0f, 1.0f);
    return length(pa - ba * h) - r;
}

__device__ float sdf_tile_frustum(vec3 p, Plane *f, uvec3 coord) {
    float d = -max_dist;
    for (int i = 0; i < 3; ++i) {
        int j = coord[i] + planes_size * i;
        d = max(d, sdf_plane(p + f[j].n * f[j].o, -f[j].n));
        d = max(d, sdf_plane(p + f[j+1].n * f[j+1].o, f[j+1].n));
    }
    return d;
}

__device__ float sdf_tile_planes(vec3 p, Plane *f, uvec3 coord) {
    float d = max_dist;
    for (int i = 0; i < 3; ++i) {
        int j = coord[i] + planes_size * i;
        d = min(d, abs(sdf_plane(p + f[j].n * f[j].o, -f[j].n)));
        d = min(d, abs(sdf_plane(p + f[j+1].n * f[j+1].o, f[j+1].n)));
    }
    return d;
}

__device__ float sdf_frustum(vec3 p, Plane *f) {
    float d = -max_dist;
    for (int i = 0; i < 3; ++i) {
        int b = i * planes_size;
        int e = (i+1) * planes_size-1;
        d = max(d, sdf_plane(p + f[b].n * f[b].o, -f[b].n));
        d = max(d, sdf_plane(p + f[e].n * f[e].o, f[e].n));
    }
    return d;
}

__device__ float sdf_lights(vec3 p, KeyValue *mortons, Light *l, int b, int e) {
    float d = max_dist;
    for (int i = b; i < e; ++i) {
        d = sdmin(d, sdf_sphere(l[mortons[i].v].p - p) - l[mortons[i].v].r);
    }
    return d;
}

__device__ float sdf_tile_lights(vec3 p, int* indices, int n, Light *lights) {
    float d = max_dist;
    for (int i = 0; i < n; ++i) {
        Light l = lights[indices[i]];
        d = sdmin(d, sdf_sphere(l.p - p) - l.r);
    }
    return d;
}

__device__ float sdf_scene_frustum(Scene s, vec3 p) {
    return sdf_frustum(p, s.f);
}

__device__ float sdf_scene_tile_lights(Scene s, vec3 p) {
    return sdf_tile_lights(p, s.indices + s.spans[s.tile_index].begin, s.spans[s.tile_index].count, s.l);
}

__device__ float sdf_scene_tile_frustum(Scene s, vec3 p) {
    return sdf_tile_frustum(p, s.f, tileIndexToCoord(s.tile_index));
}

__device__ float sdf_scene_lights(Scene s, vec3 p) {
    const int stride = 32;
    int i = abs(s.v.lights_offset) % ((s.lights_count-1)/stride+1);
    return sdf_lights(p, s.mortons, s.l, i * stride, min((i+1) * stride, s.lights_count));
}

__device__ struct Ray {
    bool hit;
    float l;
    float sgn;
};

using SdfScene = float(*)(Scene, vec3);

__device__ Ray march(Scene s, SdfScene sdf, vec3 ro, vec3 rd) {
    auto lo = 0.0f;
    for (int i = 0; i < max_it && lo < max_dist; ++i) {
        float sl = sdf(s, ro);
        float l = abs(sl);
        ro += l * rd;
        lo += l;

        if (l < min_dist)
            return Ray {true, lo, sign(sl)};
    }
    return Ray {false};
}

__device__ vec3 normal(Scene s, SdfScene sdf, vec3 p) {
    float l = sdf(s, p);
    float o = min_dist * 0.5f;
    return normalize(
        l - vec3(
            sdf(s, p - vec3(o, 0, 0)),
            sdf(s, p - vec3(0, o, 0)),
            sdf(s, p - vec3(0, 0, o))
        )
    );
}

__device__ mat3 look_at(vec3 d) {
    vec3 r = normalize(cross(d, vec3(0, 1, 0)));
    vec3 u = normalize(cross(r, d));
    return mat3(r, u, d);
}

__device__ float trace(Scene s, SdfScene sdf, vec3 ro, vec3 rd) {
    float c = 0.0f;
    for (float i = 0.0f; i < 8.0f; ++i) {

        Ray r = march(s, sdf, ro, rd);
        if (!r.hit)
            break;

        vec3 p = ro + rd * r.l;
        vec3 n = normal(s, sdf, p);
        ro = p + rd * min_dist * 10.0f;
        float front = max(0.0f, dot(rd, -n));
        if (r.l < min_dist * 1.0f) {
            i -= 0.5f;
        } else if (0.0f < front) {
            vec3 directional_light = normalize(vec3(1, 3, 2));
            float a = dot(n, directional_light);
            c += pow(1.0f-front, 5.0f)
                + 0.8f * max(0.0f, a)
                + 0.8f * pow(max(0.0f, -a), 5.0f)
                + 0.2f;
        }
    }
    return c;
}

__global__ void get_image(vec4 *c, Scene s) {
    int gtid = threadIdx.x + blockIdx.x * blockDim.x;
    vec2 uv = vec2(gtid % s.cam.res.x, gtid / s.cam.res.x) / vec2(s.cam.res) * 2.0f - 1.0f;

    // vec3 ro = cam.eye;
    // vec3 rd = look_at(normalize(cam.dir)) * normalize(vec3(uv, 1.0));
    vec3 ro = s.v.zoom * vec3(
        sin(s.v.origin.x) * cos(s.v.origin.y),
        sin(s.v.origin.y),
        cos(s.v.origin.x) * cos(s.v.origin.y)
    );
    vec3 rd = look_at(normalize(-ro)) * normalize(vec3(uv, 1.0));
    vec3 center = vec3(s.v.look_at.x, 0, s.v.look_at.y);
    ro += center;

    c[gtid] = vec4((1.0f/255.0f) * vec3(n21(uv)), 1);
    if (s.v.visible_flags & View::VisibleFlag::frustum)
        c[gtid].b += 0.8f * trace(s, &sdf_scene_frustum, ro, rd);
    if (s.v.visible_flags & View::VisibleFlag::tile_frustum)
        c[gtid].g += 0.8f * trace(s, &sdf_scene_tile_frustum, ro, rd);
    if (s.v.visible_flags & View::VisibleFlag::tile_lights)
        c[gtid].r += 0.3f * trace(s, &sdf_scene_tile_lights, ro, rd);
    if (s.v.visible_flags & View::VisibleFlag::lights)
        c[gtid].b += 0.3f * trace(s, &sdf_scene_lights, ro, rd);

    // draw look_at position
    c[gtid].g += trace(s, [] (Scene s, vec3 p) -> float {
        return length(vec3(s.v.look_at.x, 0, s.v.look_at.y) - p) - 0.1f;
    }, ro, rd);

    // gamma correction
    c[gtid] = vec4(pow(vec3(c[gtid]), vec3(1.0 / 2.2)), c[gtid].a);
}

void frustumPlanes(Plane *planes, Camera cam)
{
    const auto up = vec3(0, 1, 0);
    const auto invProj = inverse(perspective(radians(cam.proj.fov), 1.0f, cam.proj.near, cam.proj.far));
    auto verticalPlanes = planes;
    auto horizontalPlanes = verticalPlanes + planes_size;
    auto parallelPlanes = horizontalPlanes + planes_size;

    for (auto i = 0ul; i < planes_size; ++i) {
        auto x = -1.0f + 2.0f * i / grid_size;
        auto forward = normalize(vec3(invProj * vec4(x, 0, 0, 1)));
        auto normal = normalize(cross(forward, up));
        verticalPlanes[i].n = normal;
        verticalPlanes[i].o = 0.0f;
    }

    for (auto i = 0ul; i < planes_size; ++i) {
        auto y = -1.0f + 2.0f * i / grid_size;
        auto forward = normalize(vec3(invProj * vec4(0, y, 0, 1)));
        auto right = normalize(cross(forward, up));
        auto normal = normalize(cross(right, forward));
        horizontalPlanes[i].n = normal;
        horizontalPlanes[i].o = 0.0f;
    }

    for (auto i = 0ul; i < planes_size; ++i) {
        parallelPlanes[i].n = vec3(0, 0, 1);
        parallelPlanes[i].o = cam.proj.near + (cam.proj.far-cam.proj.near) * float(grid_size-i) / grid_size;
    }
}

void draw(vec4 *image, Camera cam, Span *spans, int* indices, KeyValue* mortons, Light *lights, int n, uvec3 tile_coord, View view) {
    Plane *frustum;
    hipMallocManaged(&frustum, planes_count * sizeof(Plane));
    hipDeviceSynchronize();
    frustumPlanes(frustum, cam);
    hipDeviceSynchronize();

    int w = 256;
    int b = (cam.res.x*cam.res.y-1)/w+1;
    unsigned int tile_index = tileCoordToIndex(tile_coord);
    get_image<<<b, w>>>(image, Scene {spans, indices, mortons, lights, frustum, n, tile_index, view, cam});
}

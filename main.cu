#include "hip/hip_runtime.h"
#include "sort.cu"
#include "draw.cu"
#include "assign_lights.cu"
#include "display.cu"

template <typename T>
void dump(T* a, int n, const char* label) {
    std::cout << "\t" << label << std::endl;
    for (int i = 0; i < n; ++i)
        std::cout << a[i] << ", " << std::endl;
}

std::ostream& operator<< (std::ostream& os, KeyValue a) {
    return os << "(k: " << a.k << ", v: " <<  a.v << ")";
}

std::ostream& operator<< (std::ostream& os, Light a) {
    return os << "(p: [" << a.p.x << ", " << a.p.y << ", " << a.p.z << "], r: " << a.r << ")";
}

std::ostream& operator<< (std::ostream& os, Span a) {
    return os << "(begin: " << a.begin << ", " << ", count: " << a.count << ")";
}

std::ostream& operator<< (std::ostream& os, Aabb a) {
    return os << "(back_left_bot: [" << a.back_left_bot.x << ", " << a.back_left_bot.y << ", " << a.back_left_bot.z << "], front_right_top: [" << a.front_right_top.x << ", " << a.front_right_top.y << ", " << a.front_right_top.z << "])";
}

int main([[maybe_unused]] int argc, [[maybe_unused]] char* argv[])
{
    int elements = 8*8*32;

    if (argc == 2) {
        elements = 1<<std::stoi(argv[1]);
        elements = max(32, elements);
    }
    std::cout << "elements: " << elements << std::endl;

    int w = 256;
    int b = (elements-1)/w+1;
    int b32 = (elements-1)/32+1;

    Camera cam {
        .eye = {0, 0, 5},
        .dir = {0, 0, -1},
        .res = {960, 960},
        .proj = Perspective {90.0f, 1.0f, 32.0f},
    };

    Light *lights;
    KeyValue *mortons;
    Aabb *aabbs;
    Aabb *aabbs32;
    glm::vec4 *image;
    int *indices;
    const int indices_capacity = 1024*1024;
    int *indices_size;
    Span *spans;
    hipMallocManaged(&lights, elements * sizeof(Light));
    hipMallocManaged(&mortons, elements * sizeof(KeyValue));
    hipMallocManaged(&aabbs32, ((elements-1)/w+1) * sizeof(Aabb));
    hipMallocManaged(&aabbs, elements * sizeof(Aabb));
    SortByKey h_sortp;
    hipMemcpyFromSymbol(&h_sortp, HIP_SYMBOL(d_sortp), sizeof(SortByKey));
    hipMallocManaged(&image, (cam.res.x * cam.res.y) * sizeof(glm::vec4));
    hipMallocManaged(&indices, indices_capacity * sizeof(int));
    hipMallocManaged(&indices_size, sizeof(int));
    hipMallocManaged(&spans, tiles_count * sizeof(Span));

    hipDeviceSynchronize();
    init_lights<<<b, w>>>(lights, elements);

    hipDeviceSynchronize();
    get_mortons<<<b, w>>>(lights, mortons, elements, cam.proj);

#ifdef OPT_BVH
    hipDeviceSynchronize();
    sort(&mortons, elements, h_sortp);
#endif

    hipDeviceSynchronize();
    get_aabbs<<<b, w>>>(mortons, lights, aabbs, elements, cam.proj);

#ifdef OPT_BVH
    hipDeviceSynchronize();
    reduce_aabbs<<<b, w>>>(aabbs, aabbs32);
#endif

    hipDeviceSynchronize();
    assign_lights<<<tiles_count, w>>>(mortons, aabbs32, aabbs, elements, spans, indices, indices_size, indices_capacity);

    hipDeviceSynchronize();
    // dump(mortons, elements, "mortons");
    // dump(aabbs32, elements/32, "aabss32");
    // dump(aabbs, elements, "aabss");
    // dump(spans, tiles_count, "spans");

    auto update = [&] (uvec3 tile_coord, View view) {
        hipDeviceSynchronize();
        draw(image, cam, spans, indices, mortons, lights, elements, tile_coord, view);

        hipDeviceSynchronize();
        return image;
    };

    display(cam.res, update);

    hipFree(lights);
    hipFree(mortons);
    hipFree(aabbs);
    hipFree(aabbs32);
    hipFree(image);
    hipFree(indices);
    hipFree(indices_size);
    hipFree(spans);

    return 0;
}

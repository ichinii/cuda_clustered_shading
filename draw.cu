#include "hip/hip_runtime.h"
#pragma once

#include "common.cu"

#define max_dist 200.0f
#define min_dist 0.01f
#define max_it 1000
#define planes_size (grid_size + 1u)
#define planes_count (planes_size + planes_size + planes_size)

__device__ struct Scene {
    Span *spans;
    int *indices;
    KeyValue *mortons;
    Light *l;
    Plane *f;
    int lights_count;
    unsigned int tile_index;
    View v;
    Camera cam;
};

__device__ float sdmin(float a, float b) {
    return abs(a) < abs(b) ? a : b;
}

__device__ float sdf_plane(vec3 p, vec3 n) {
    return dot(p, n);
}

__device__ float sdf_sphere(vec3 p) {
    return length(p);
}

// thanks Inigo Quilez (https://iquilezles.org/articles/distfunctions/)
__device__ float sdf_capsule(vec3 p, vec3 a, vec3 b, float r) {
    vec3 pa = p - a;
    vec3 ba = b - a;
    float h = clamp(dot(pa, ba) / dot(ba, ba), 0.0f, 1.0f);
    return length(pa - ba * h) - r;
}

__device__ float sdf_tile_frustum(vec3 p, Plane *f, uvec3 coord) {
    float d = -max_dist;
    for (int i = 0; i < 3; ++i) {
        int j = coord[i] + planes_size * i;
        d = max(d, sdf_plane(p + f[j].n * f[j].o, -f[j].n));
        d = max(d, sdf_plane(p + f[j+1].n * f[j+1].o, f[j+1].n));
    }
    return d;
}

__device__ float sdf_tile_planes(vec3 p, Plane *f, uvec3 coord) {
    float d = max_dist;
    for (int i = 0; i < 3; ++i) {
        int j = coord[i] + planes_size * i;
        d = min(d, abs(sdf_plane(p + f[j].n * f[j].o, -f[j].n)));
        d = min(d, abs(sdf_plane(p + f[j+1].n * f[j+1].o, f[j+1].n)));
    }
    return d;
}

__device__ float sdf_frustum(vec3 p, Plane *f) {
    float d = -max_dist;
    for (int i = 0; i < 3; ++i) {
        int b = i * planes_size;
        int e = (i+1) * planes_size-1;
        d = max(d, sdf_plane(p + f[b].n * f[b].o, -f[b].n));
        d = max(d, sdf_plane(p + f[e].n * f[e].o, f[e].n));
    }
    return d;
}

__device__ float sdf_lights(vec3 p, KeyValue *mortons, Light *l, int b, int e) {
    float d = max_dist;
    for (int i = b; i < e; ++i) {
        d = sdmin(d, sdf_sphere(l[mortons[i].v].p - p) - l[mortons[i].v].r);
    }
    return d;
}

__device__ float sdf_tile_lights(vec3 p, int* indices, int n, Light *lights) {
    float d = max_dist;
    for (int i = 0; i < n; ++i) {
        Light l = lights[indices[i]];
        d = sdmin(d, sdf_sphere(l.p - p) - l.r);
    }
    return d;
}

__device__ float sdf_scene_frustum(Scene s, vec3 p) {
    return sdf_frustum(p, s.f);
}

__device__ float sdf_scene_tile_lights(Scene s, vec3 p) {
    return sdf_tile_lights(p, s.indices + s.spans[s.tile_index].begin, s.spans[s.tile_index].count, s.l);
}

__device__ float sdf_scene_tile_frustum(Scene s, vec3 p) {
    return sdf_tile_frustum(p, s.f, tileIndexToCoord(s.tile_index));
}

__device__ float sdf_scene_tile_helper_lines(Scene s, vec3 p) {
    float d = max_dist;
    float r = 0.1f;
    vec3 coord = vec3(tileIndexToCoord(s.tile_index));
    vec3 a = coord / float(grid_size) * 2.0f - 1.0f;        // back left bot
    vec3 b = (coord+1.0f) / float(grid_size) * 2.0f - 1.0f; // front right top
    a.z -= s.cam.proj.near;
    b.z -= s.cam.proj.near;
    d = sdmin(d, sdf_capsule(p, vec3(a.x, a.y, 0), vec3(a.x, a.y, 1), r));
    d = sdmin(d, sdf_capsule(p, vec3(b.x, a.y, 0), vec3(b.x, a.y, 1), r));
    d = sdmin(d, sdf_capsule(p, vec3(a.x, b.y, 0), vec3(a.x, b.y, 1), r));
    d = sdmin(d, sdf_capsule(p, vec3(b.x, b.y, 0), vec3(b.x, b.y, 1), r));
    return d;
}

__device__ float sdf_scene_lights(Scene s, vec3 p) {
    const int stride = 32;
    int i = s.v.lights_offset % ((s.lights_count-1)/stride+1);
    return sdf_lights(p, s.mortons, s.l, i * stride, min((i+1) * stride, s.lights_count));
}

__device__ struct Ray {
    bool hit;
    float l;
    float sgn;
};

__device__ float sdf_frustum_depth(Scene s, vec3 p) {
    float d = max_dist;
    d = sdmin(d, sdf_plane(p + s.cam.proj.near, vec3(0, 0, 1)));
    d = sdmin(d, sdf_plane(p + s.cam.proj.far, vec3(0, 0, -1)));
    return d;
}

using SdfScene = float(*)(Scene, vec3);

__device__ Ray march(Scene s, SdfScene sdf, vec3 ro, vec3 rd) {
    auto lo = 0.0f;
    for (int i = 0; i < max_it && lo < max_dist; ++i) {
        float sl = sdf(s, ro);
        float l = abs(sl);
        ro += l * rd;
        lo += l;

        if (l < min_dist)
            return Ray {true, lo, sign(sl)};
    }
    return Ray {false};
}

__device__ vec3 normal(Scene s, SdfScene sdf, vec3 p) {
    float l = sdf(s, p);
    float o = min_dist * 0.5f;
    return normalize(
        l - vec3(
            sdf(s, p - vec3(o, 0, 0)),
            sdf(s, p - vec3(0, o, 0)),
            sdf(s, p - vec3(0, 0, o))
        )
    );
}

__device__ mat3 look_at(vec3 d) {
    vec3 r = normalize(cross(d, vec3(0, 1, 0)));
    vec3 u = normalize(cross(r, d));
    return mat3(r, u, d);
}

__device__ float trace(Scene s, SdfScene sdf, vec3 ro, vec3 rd) {
    float c = 0.0f;
    for (float i = 0.0f; i < 8.0f; ++i) {

        Ray r = march(s, sdf, ro, rd);
        if (!r.hit)
            break;

        vec3 p = ro + rd * r.l;
        vec3 n = normal(s, sdf, p);
        ro = p + rd * min_dist * 10.0f;
        float front = max(0.0f, dot(rd, -n));
        if (r.l < min_dist * 1.0f) {
            i -= 0.5f;
        } else if (0.0f < front) {
            c += (0.7f + 0.3f * front)
                * (0.7f + 0.3f * dot(n, normalize(vec3(1, 3, 2))));
        }
    }
    return c;
}

__global__ void get_image(vec4 *c, Scene s) {
    int gtid = threadIdx.x + blockIdx.x * blockDim.x;
    vec2 uv = vec2(gtid % s.cam.res.x, gtid / s.cam.res.x) / vec2(s.cam.res) * 2.0f - 1.0f;

    // vec3 ro = cam.eye;
    // vec3 rd = look_at(normalize(cam.dir)) * normalize(vec3(uv, 1.0));
    vec3 ro = s.v.zoom * vec3(
        sin(s.v.origin.x) * cos(s.v.origin.y),
        sin(s.v.origin.y),
        cos(s.v.origin.x) * cos(s.v.origin.y)
    );
    vec3 rd = look_at(normalize(-ro)) * normalize(vec3(uv, 1.0));
    vec3 center = vec3(s.v.look_at.x, 0, s.v.look_at.y);
    ro += center;

    c[gtid] = vec4((1.0f/255.0f) * vec3(n21(uv)), 1);
    c[gtid].b += 0.8f * trace(s, &sdf_scene_frustum, ro, rd);
    c[gtid].g += 0.8f * trace(s, &sdf_scene_tile_frustum, ro, rd);
    c[gtid].r += 0.3f * trace(s, &sdf_scene_tile_lights, ro, rd);
    // c[gtid].g += trace(s, &sdf_scene_tile_helper_lines, ro, rd);
    // c[gtid].r += trace(s, &sdf_frustum_depth, ro, rd);
    float lights = 0.5f * trace(s, &sdf_scene_lights, ro, rd);
    c[gtid].b += lights;

    // draw look_at position
    c[gtid].g += trace(s, [] (Scene s, vec3 p) -> float {
        return length(vec3(s.v.look_at.x, 0, s.v.look_at.y) - p) - 0.15f;
    }, ro, rd);
}

void frustumPlanes(Plane *planes, Camera cam)
{
    const auto up = vec3(0, 1, 0);
    const auto invProj = inverse(perspective(radians(cam.proj.fov), 1.0f, cam.proj.near, cam.proj.far));
    auto verticalPlanes = planes;
    auto horizontalPlanes = verticalPlanes + planes_size;
    auto parallelPlanes = horizontalPlanes + planes_size;

    for (auto i = 0ul; i < planes_size; ++i) {
        auto x = -1.0f + 2.0f * i / grid_size;
        auto forward = normalize(vec3(invProj * vec4(x, 0, 0, 1)));
        auto normal = normalize(cross(forward, up));
        verticalPlanes[i].n = normal;
        verticalPlanes[i].o = 0.0f;
    }

    for (auto i = 0ul; i < planes_size; ++i) {
        auto y = -1.0f + 2.0f * i / grid_size;
        auto forward = normalize(vec3(invProj * vec4(0, y, 0, 1)));
        auto right = normalize(cross(forward, up));
        auto normal = normalize(cross(right, forward));
        horizontalPlanes[i].n = normal;
        horizontalPlanes[i].o = 0.0f;
    }

    for (auto i = 0ul; i < planes_size; ++i) {
        parallelPlanes[i].n = vec3(0, 0, 1);
        parallelPlanes[i].o = cam.proj.near + (cam.proj.far-cam.proj.near) * float(grid_size-i) / grid_size;
    }
}

void draw(vec4 *image, Camera cam, Span *spans, int* indices, KeyValue* mortons, Light *lights, int n, uvec3 tile_coord, View view) {
    Plane *frustum;
    hipMallocManaged(&frustum, planes_count * sizeof(Plane));
    hipDeviceSynchronize();
    frustumPlanes(frustum, cam);
    hipDeviceSynchronize();

    int w = 256;
    int b = (cam.res.x*cam.res.y-1)/w+1;
    unsigned int tile_index = tileCoordToIndex(tile_coord);
    get_image<<<b, w>>>(image, Scene {spans, indices, mortons, lights, frustum, n, tile_index, view, cam});
}

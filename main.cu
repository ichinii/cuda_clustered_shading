#include "hip/hip_runtime.h"
#include "sort.cu"
#include "draw.cu"
#include "assign_lights.cu"
#include "display.cu"

int main([[maybe_unused]] int argc, [[maybe_unused]] char* argv[])
{
    // number of light sources
    int elements = 8*8*32;

    if (argc == 2) {
        elements = 1<<std::stoi(argv[1]);
        elements = max(32, elements);
    }
    std::cout << "elements: " << elements << std::endl;

    int w = 256;
    int b = (elements-1)/w+1;
    int b32 = (elements-1)/32+1;

    Camera cam {
        .eye = {0, 0, 5},
        .dir = {0, 0, -1},
        .res = {960, 960},
        .proj = Perspective {90.0f, 1.0f, 32.0f},
    };

    Light *lights;
    KeyValue *mortons;
    Aabb *aabbs;
    Aabb *aabbs32;
    glm::vec4 *image;
    int *indices;
    const int indices_capacity = 1024*1024;
    int *indices_size;
    Span *spans;
    hipMallocManaged(&lights, elements * sizeof(Light));
    hipMallocManaged(&mortons, elements * sizeof(KeyValue));
    hipMallocManaged(&aabbs32, ((elements-1)/w+1) * sizeof(Aabb));
    hipMallocManaged(&aabbs, elements * sizeof(Aabb));
    SortByKey h_sortp;
    hipMemcpyFromSymbol(&h_sortp, HIP_SYMBOL(d_sortp), sizeof(SortByKey));
    hipMallocManaged(&image, (cam.res.x * cam.res.y) * sizeof(glm::vec4));
    hipMallocManaged(&indices, indices_capacity * sizeof(int));
    hipMallocManaged(&indices_size, sizeof(int));
    hipMallocManaged(&spans, tiles_count * sizeof(Span));

    hipDeviceSynchronize();
    init_lights<<<b, w>>>(lights, elements);

    hipDeviceSynchronize();
    get_mortons<<<b, w>>>(lights, mortons, elements, cam.proj);

#ifdef OPT_BVH
    hipDeviceSynchronize();
    sort(&mortons, elements, h_sortp);
#endif

    hipDeviceSynchronize();
    get_aabbs<<<b, w>>>(mortons, lights, aabbs, elements, cam.proj);

#ifdef OPT_BVH
    hipDeviceSynchronize();
    reduce_aabbs<<<b, w>>>(aabbs, aabbs32);
#endif

    hipDeviceSynchronize();
    assign_lights<<<tiles_count, w>>>(mortons, aabbs32, aabbs, elements, spans, indices, indices_size, indices_capacity);

    hipDeviceSynchronize();
    // dump(mortons, elements, "mortons");
    // dump(aabbs32, elements/32, "aabss32");
    // dump(aabbs, elements, "aabss");
    // dump(spans, tiles_count, "spans");

    auto update = [&] (uvec3 tile_coord, View view) {
        hipDeviceSynchronize();
        draw(image, cam, spans, indices, mortons, lights, elements, tile_coord, view);

        hipDeviceSynchronize();
        return image;
    };

    display(cam.res, update);

    hipFree(lights);
    hipFree(mortons);
    hipFree(aabbs);
    hipFree(aabbs32);
    hipFree(image);
    hipFree(indices);
    hipFree(indices_size);
    hipFree(spans);

    return 0;
}
